#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <sys/stat.h>
#include <sys/types.h>
#include <sys/mman.h>
#include <fcntl.h>
#include <unistd.h>
#include <hip/hip_runtime.h>


__global__ void cuda_hello(){
    printf("Hello World from GPU!\n");
}
int main() {
    cuda_hello<<<1,1>>>(); 
    hipDeviceSynchronize();
    return 0;
}
