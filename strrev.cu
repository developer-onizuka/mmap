#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <sys/stat.h>
#include <sys/types.h>
#include <sys/mman.h>
#include <fcntl.h>
#include <unistd.h>
#include <hip/hip_runtime.h>


#define INIT_BUFSIZE 4096

__global__ void strrev(char *str) {
	int size = 0;
        while (str[size] != '\0') {
                size++;
        }
	for(int i=0;i<size/2;++i) {
		char t = str[i];
		str[i] = str[size-1-i];
		str[size-1-i] = t;
	}
}

int main(int argc, char *argv[])
{
	int fd;
	char *system_buf;
	char *gpumem_buf;
	int buf_size = INIT_BUFSIZE;
	system_buf = (char*)malloc(buf_size);
	hipMalloc((void**)&gpumem_buf, buf_size);

	fd = open(argv[1], O_RDWR);
	pread(fd, system_buf, buf_size, 0);
	hipMemcpy(gpumem_buf, system_buf, buf_size, hipMemcpyHostToDevice);
	strrev<<<1,1>>>(gpumem_buf);

	hipMemcpy(system_buf, gpumem_buf, buf_size, hipMemcpyDeviceToHost);
	printf("%s: %s\n", argv[1], system_buf);

	hipFree(gpumem_buf);
	free(system_buf);

	close(fd);
}
